char knn(Point* all_points, char* all_labels, int total_points, int k, Point to_evaluate){
    float *dev_all_distances;
    char *dev_label_distances;
    Point *dev_all_points;
    char *dev_all_labels;
    hipError_t err;

    printf("antes do malloc e tamanho total: %d\n", total_points);

    err = hipMalloc((void**)&dev_all_points, total_points * sizeof(Point));
    if (err != hipSuccess) {
        printf("Erro ao alocar dev_all_points: %s\n", hipGetErrorString(err));
        return -1;
    }
    printf("aloquei 1\n");

    err = hipMalloc((void**)&dev_all_labels, total_points * sizeof(char));
    if (err != hipSuccess) {
        printf("Erro ao alocar dev_all_labels: %s\n", hipGetErrorString(err));
        return -1;
    }
    printf("aloquei 2\n");

    err = hipMalloc((void**)&dev_all_distances, total_points * sizeof(float));
    if (err != hipSuccess) {
        printf("Erro ao alocar dev_all_distances: %s\n", hipGetErrorString(err));
        return -1;
    }
    printf("aloquei 3\n");

    err = hipMalloc((void**)&dev_label_distances, total_points * sizeof(char));
    if (err != hipSuccess) {
        printf("Erro ao alocar dev_label_distances: %s\n", hipGetErrorString(err));
        return -1;
    }
    printf("aloquei 4\n");

    printf("antes do memcpy\n");

    err = hipMemcpy(dev_all_points, all_points, total_points * sizeof(Point), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Erro ao copiar dev_all_points: %s\n", hipGetErrorString(err));
        return -1;
    }

    err = hipMemcpy(dev_all_labels, all_labels, total_points * sizeof(char), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Erro ao copiar dev_all_labels: %s\n", hipGetErrorString(err));
        return -1;
    }

    printf("antes do dim3\n");

    dim3 threads(128);
    dim3 blocks((total_points + (threads.x - 1)) / threads.x);

    printf("antes do kernel\n");
    calcular_distancia<<<blocks, threads>>>(dev_all_points, dev_all_labels, dev_all_distances, dev_label_distances, total_points, k, to_evaluate);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Erro no kernel calcular_distancia: %s\n", hipGetErrorString(err));
        return -1;
    }

    printf("terminei as distancias\n");

    float* dev_sorted_distances;
    char* dev_sorted_labels;

    err = hipMalloc(&dev_sorted_distances, total_points * sizeof(float));
    if (err != hipSuccess) {
        printf("Erro ao alocar dev_sorted_distances: %s\n", hipGetErrorString(err));
        return -1;
    }

    err = hipMalloc(&dev_sorted_labels, total_points * sizeof(char));
    if (err != hipSuccess) {
        printf("Erro ao alocar dev_sorted_labels: %s\n", hipGetErrorString(err));
        return -1;
    }

    ordenar_distancias(dev_all_distances, dev_sorted_distances, dev_label_distances, dev_sorted_labels, total_points);

    char* k_menores_labels = (char*)malloc(k * sizeof(char));
    hipMemcpy(k_menores_labels, dev_sorted_labels, k * sizeof(char), hipMemcpyDeviceToHost);

    char resultado = calcular_mais_frequente(k_menores_labels, k);
    return resultado;
}
